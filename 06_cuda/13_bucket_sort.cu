
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void sort(int* key, int* bucket)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = 0;
    if (i < 5)
    {
        bucket[i] = 0;
    }
    __syncthreads();
    atomicAdd(&bucket[key[i]], 1);
    __syncthreads();
    if (i < 5)
    {
        for (int k = i; k > 0; --k)
            j += bucket[k - 1];
        for (; bucket[i] > 0; bucket[i]--) {
            key[j++] = i;
        }
    }
}

int main() {
    const int N = 50;
    int n = 50;
    int range = 5;
    int* key, * bucket;
    hipMallocManaged(&key, n * sizeof(int));
    hipMallocManaged(&bucket, range * sizeof(int));
    //std::vector<int> key(n);
    for (int i = 0; i < n; i++) {
        key[i] = rand() % range;
        printf("%d ", key[i]);
    }
    printf("\n");

    sort <<<1, N>>> (key, bucket);
    hipDeviceSynchronize();
    //std::vector<int> bucket(range);
    //for (int i = 0; i < range; i++) {
    //    bucket[i] = 0;
    //}
    //for (int i = 0; i < n; i++) {
    //    bucket[key[i]]++;
    //}
    //for (int i = 0, j = 0; i < range; i++) {
    //    for (; bucket[i] > 0; bucket[i]--) {
    //        key[j++] = i;
    //    }
    //}
    for (int i = 0; i < n; i++) {
        printf("%d ", key[i]);
    }
    printf("\n");
    hipFree(key);
    hipFree(bucket);
}